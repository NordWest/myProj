#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Example of integrating CUDA functions into an existing
 * application / framework.
 * Host part of the device code.
 * Compiled with Cuda compiler.
 */

// System includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif

#define EKV 0.409092804
#define ka 0.017202098955
#define CAU 173.144632685

#include "rada.h"

extern ever_params *eparam;
extern int nofzbody;
extern double *mass;

////////////////////////////////////////////////////////////////////////////////
// declaration, forward

extern "C" void computeGold(char *reference, char *idata, const unsigned int len);
extern "C" void computeGold2(int2 *reference, int2 *idata, const unsigned int len);

///////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_odata  memory to process (in and out)
///////////////////////////////////////////////////////////////////////////////
__global__ void kernel(int *g_data)
{
    // write data to global memory
    const unsigned int tid = threadIdx.x;
    int data = g_data[tid];

    // use integer arithmetic to process all four bytes with one thread
    // this serializes the execution, but is the simplest solutions to avoid
    // bank conflicts for this very low number of threads
    // in general it is more efficient to process each byte by a separate thread,
    // to avoid bank conflicts the access pattern should be
    // g_data[4 * wtid + wid], where wtid is the thread id within the half warp
    // and wid is the warp id
    // see also the programming guide for a more in depth discussion.
    g_data[tid] = ((((data <<  0) >> 24) - 10) << 24)
                  | ((((data <<  8) >> 24) - 10) << 16)
                  | ((((data << 16) >> 24) - 10) <<  8)
                  | ((((data << 24) >> 24) - 10) <<  0);
}


///////////////////////////////////////////////////////////////////////////////
//! Demonstration that int2 data can be used in the cpp code
//! @param g_odata  memory to process (in and out)
///////////////////////////////////////////////////////////////////////////////
__global__ void
kernel2(int2 *g_data)
{
    // write data to global memory
    const unsigned int tid = threadIdx.x;
    int2 data = g_data[tid];

    // use integer arithmetic to process all four bytes with one thread
    // this serializes the execution, but is the simplest solutions to avoid
    // bank conflicts for this very low number of threads
    // in general it is more efficient to process each byte by a separate thread,
    // to avoid bank conflicts the access pattern should be
    // g_data[4 * wtid + wid], where wtid is the thread id within the half warp
    // and wid is the warp id
    // see also the programming guide for a more in depth discussion.
    g_data[tid].x = data.x - data.y;
}

extern "C" __global__ void VecAdd_kernel(const float *A, const float *B, float *C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N)
        C[i] = A[i] + B[i];
}

__global__ void
vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Entry point for Cuda functionality on host side
//! @param argc  command line argument count
//! @param argv  command line arguments
//! @param data  data to process on the device
//! @param len   len of \a data
////////////////////////////////////////////////////////////////////////////////
extern "C" bool
runTest(const int argc, const char **argv, char *data, int2 *data_int2, unsigned int len)
{
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    findCudaDevice(argc, (const char **)argv);

    const unsigned int num_threads = len / 4;
    assert(0 == (len % 4));
    const unsigned int mem_size = sizeof(char) * len;
    const unsigned int mem_size_int2 = sizeof(int2) * len;

    // allocate device memory
    char *d_data;
    checkCudaErrors(hipMalloc((void **) &d_data, mem_size));
    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_data, data, mem_size,
                               hipMemcpyHostToDevice));
    // allocate device memory for int2 version
    int2 *d_data_int2;
    checkCudaErrors(hipMalloc((void **) &d_data_int2, mem_size_int2));
    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_data_int2, data_int2, mem_size_int2,
                               hipMemcpyHostToDevice));

    // setup execution parameters
    dim3 grid(1, 1, 1);
    dim3 threads(num_threads, 1, 1);
    dim3 threads2(len, 1, 1); // more threads needed fir separate int2 version
    // execute the kernel
    kernel<<< grid, threads >>>((int *) d_data);
    kernel2<<< grid, threads2 >>>(d_data_int2);

    // check if kernel execution generated and error
    getLastCudaError("Kernel execution failed");

    // compute reference solutions
    char *reference = (char *) malloc(mem_size);
    computeGold(reference, data, len);
    int2 *reference2 = (int2 *) malloc(mem_size_int2);
    computeGold2(reference2, data_int2, len);

    // copy results from device to host
    checkCudaErrors(hipMemcpy(data, d_data, mem_size,
                               hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(data_int2, d_data_int2, mem_size_int2,
                               hipMemcpyDeviceToHost));

    // check result
    bool success = true;

    for (unsigned int i = 0; i < len; i++)
    {
        if (reference[i] != data[i] ||
            reference2[i].x != data_int2[i].x ||
            reference2[i].y != data_int2[i].y)
        {
            success = false;
        }
    }

    // cleanup memory
    checkCudaErrors(hipFree(d_data));
    checkCudaErrors(hipFree(d_data_int2));
    free(reference);
    free(reference2);

    return success;
}


extern "C" bool
runSumm(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host input vector B
    float *h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device input vector A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("Test PASSED\n");

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Reset the device and exit
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}

__global__ void test_force_GN_kernel(double X[], double V[], double F[], int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    int teloi=i/3;
    if (i < numElements)
    {

        F[i] = X[i]*V[i];
    }
}

extern "C" void test_force_GN_CU(double X[], double V[], double F[])
{
  int iNum = nofzbody;
  int Ni = iNum*3;
  int i;

  hipError_t err = hipSuccess;

  printf("nofzbody: %d\nNi: %d\n", nofzbody, Ni);

  // Allocate the device input vector A
  size_t size = Ni * sizeof(double);
  double *d_X = NULL;
  err = hipMalloc((void **)&d_X, size);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate device vector X (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  // Allocate the device input vector B
  double *d_V = NULL;
  err = hipMalloc((void **)&d_V, size);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate device vector V (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  // Allocate the device output vector C
  double *d_F = NULL;
  err = hipMalloc((void **)&d_F, size);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate device vector F (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  double *d_Mass = NULL;
  err = hipMalloc((void **)&d_Mass, size);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate device vector Mass (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  printf("Copy input data X from the host memory to the CUDA device\n");
  err = hipMemcpy(d_X, X, size, hipMemcpyHostToDevice);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy vector X from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  err = hipMemcpy(d_V, V, size, hipMemcpyHostToDevice);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy vector V from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  err = hipMemcpy(d_F, F, size, hipMemcpyHostToDevice);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy vector V from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  err = hipMemcpy(d_Mass, mass, size, hipMemcpyHostToDevice);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy vector V from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  int numBlocks = 1;
//  dim3 threadsPerBlock(Ni, Ni);
  test_force_GN_kernel<<<numBlocks, Ni>>>(d_X, d_V, d_F, Ni);

  //double *FS = new double[iNum];

  for(i=0;i<iNum;i++)
  {
      printf("F[%d]: %f ?= %f\n", i, d_F[i], X[i]*V[i]);

  }

  // Free device global memory
  err = hipFree(d_X);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to free device vector X (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
  err = hipFree(d_V);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to free device vector V (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
  err = hipFree(d_F);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  err = hipFree(d_Mass);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to free device vector Mass (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}
