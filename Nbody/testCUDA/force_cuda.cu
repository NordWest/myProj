#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Example of integrating CUDA functions into an existing
 * application / framework.
 * Host part of the device code.
 * Compiled with Cuda compiler.
 */

// System includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include "rada.h"
//#include <dele.h>

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif

#define EKV 0.409092804
#define ka 0.017202098955
#define CAU 173.144632685


extern ever_params *eparam;
extern int nofzbody;
extern double *mass;

extern "C" double dist3(double X0[], double X1[]);
extern "C" double norm3(double *v);

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
__global__ void force_GN_kernel(double X[], double V[], double F[], int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        F[i] = X[i]*V[i];
    }
}

extern "C" void force_GN_CU(double X[], double V[], double F[])
{
  int iNum = nofzbody;
  int Ni = iNum*3;

  //printf("nofzbody: %d\n", nofzbody);

  for(int teloi=0; teloi<iNum; teloi++)
  {
      int i=teloi*3;
      double Ri = norm3(&X[i]);

      if(Ri>(eparam->vout))
      {
          printf("WARN!!!! V OUT!!!!\n");
          printf("Ri[%d]: %f > %f\n", teloi, Ri, eparam->vout);
          exit(1);
      }

      double massI = mass[teloi];//0.0;
      //printf("massI: %e\n", massI);
      if(massI<0)massI=0;
      for(int komp=0; komp<3; komp++)
      {
          double res0, res1;
              res0 = res1 = 0.0;
//                         #pragma omp parallel for reduction(+:res0)
              for(int teloj=0; teloj<iNum; teloj++)
              {
                 int j=teloj*3;
                 double massJ = mass[teloj];
                 if(teloi!=teloj&&massJ>0)
                 {
                    double Rij = dist3(&X[i], &X[j]);
                    double Rj = norm3(&X[j]);

                    if(Rij<eparam->col)
                    {

                        printf("teloi= %d\tteloj= %d\n", teloi, teloj);
                        printf("Xi: %f\t%f\t%f\n", X[i], X[i+1], X[i+2]);
                        printf("Xj: %f\t%f\t%f\n", X[j], X[j+1], X[j+2]);
                        printf("Rij= %f\n", Rij);
                        printf("WARN!!!! CRASH!!!!\n");
                        exit(1);
                    }



                    res0 += massJ*((X[j+komp] - X[i+komp])/(pow(Rij,3)) - X[j+komp]/(pow(Rj, 3)));

                 }

              }

              res1 = -((1.0 + massI)*X[i+komp])/(pow(Ri, 3));


              F[i+komp] = ka*ka*(res0+res1);

          }
  }
}

////////////////////////////////////////////////////////////////////////////////
//! Entry point for Cuda functionality on host side
//! @param argc  command line argument count
//! @param argv  command line arguments
//! @param data  data to process on the device
//! @param len   len of \a data
////////////////////////////////////////////////////////////////////////////////
extern "C" bool
Test(const int argc, const char **argv, char *data, int2 *data_int2, unsigned int len)
{
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    findCudaDevice(argc, (const char **)argv);

    const unsigned int num_threads = len / 4;
    assert(0 == (len % 4));
    const unsigned int mem_size = sizeof(char) * len;
    const unsigned int mem_size_int2 = sizeof(int2) * len;

    // allocate device memory
    char *d_data;
    checkCudaErrors(hipMalloc((void **) &d_data, mem_size));
    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_data, data, mem_size,
                               hipMemcpyHostToDevice));
    // allocate device memory for int2 version
    int2 *d_data_int2;
    checkCudaErrors(hipMalloc((void **) &d_data_int2, mem_size_int2));
    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_data_int2, data_int2, mem_size_int2,
                               hipMemcpyHostToDevice));

    // setup execution parameters
    dim3 grid(1, 1, 1);
    dim3 threads(num_threads, 1, 1);
    dim3 threads2(len, 1, 1); // more threads needed fir separate int2 version
    // execute the kernel
    //kernel<<< grid, threads >>>((int *) d_data);
    //kernel2<<< grid, threads2 >>>(d_data_int2);

    // check if kernel execution generated and error
    getLastCudaError("Kernel execution failed");

    // compute reference solutions
    char *reference = (char *) malloc(mem_size);
    //computeGold(reference, data, len);
    int2 *reference2 = (int2 *) malloc(mem_size_int2);
    //computeGold2(reference2, data_int2, len);

    // copy results from device to host
    checkCudaErrors(hipMemcpy(data, d_data, mem_size,
                               hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(data_int2, d_data_int2, mem_size_int2,
                               hipMemcpyDeviceToHost));

    // check result
    bool success = true;

    for (unsigned int i = 0; i < len; i++)
    {
        if (reference[i] != data[i] ||
            reference2[i].x != data_int2[i].x ||
            reference2[i].y != data_int2[i].y)
        {
            success = false;
        }
    }

    // cleanup memory
    checkCudaErrors(hipFree(d_data));
    checkCudaErrors(hipFree(d_data_int2));
    free(reference);
    free(reference2);

    return success;
}


extern "C" bool
tSumm(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host input vector B
    float *h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device input vector A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    //vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("Test PASSED\n");

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Reset the device and exit
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}
